#include "hip/hip_runtime.h"
/*
* Calculating the Levenshtein distance between two strings
* This is used to find possible substitutions and spelling mistakes
* Author: Dries Nuttin
*/

#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <limits>

using namespace std;

int levenshtein(const std::string &s1, int string_length1, const std::string &s2, int string_length2);
void findClosestWords(const std::string &userInput, const std::string &dictionaryFilePath);

int main(int argc, char *argv[])
{
    std::string userInput;
    cout << "Enter a word: ";
    cin >> userInput;
    __managed__ vector<std::string> dictionary;
    std::string word;
    ifstream dictionaryFile(argv[1]);

    if (!dictionaryFile.is_open()) {
        cerr << "Failed to open dictionary.txt" << endl;
        return 1;
    }

    ifstream dictionaryFile(argv[1]);

    while (getline(dictionaryFile, word)){
        dictionary.push_back(word);
    }

    findClosestWords<<<1, 10>>>(userInput, dictionary);
    hipDeviceSynchronize();
    dictionaryFile.close();

    return 0;
}

__global__ void findClosestWords(const std::string &userInput, vector<std::string> Dictionary)
{
    
    
    std::string word;
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (tid >= Dictionary.size()) {
        return;
    }
    vector<string> closestWords;
    int minDistance = numeric_limits<int>::max();
        
    Dictionary[tid] = word;
     
        int distance = levenshtein(userInput, userInput.length(), word, word.length());
        if(distance == 0){
            std::cout << endl << "This word is spelled correctly"<< endl;
            return;
        }
        else if (distance < minDistance) {
            minDistance = distance;
            closestWords.clear();
            closestWords.push_back(word);
        } else if (distance == minDistance) {
            closestWords.push_back(word);
        }
    

    std::cout << endl << "Closest word(s) to '" << userInput << "' with a distance of " << minDistance << ":" << endl;
    for (const auto &closestWord : closestWords) {
        std::cout << closestWord << endl;
    }
    

}

int levenshtein(const std::string &s1, int string_length1, const std::string &s2, int string_length2)
{
    int sub, insert, del;
    // Check if the string is empty or not, if it is empty it would require the length of the other string as the amount of deletions to become the first string.
    if (string_length1 == 0)
    {
        return string_length2;
    }
    if (string_length2 == 0)
    {
        return string_length1;
    }

    // If the last letter is the same for both strings, we can skip this as there is no operation needed
    if (s1[string_length1 - 1] == s2[string_length2 - 1])
    {
        return levenshtein(s1, string_length1 - 1, s2, string_length2 - 1);
    }

    // Going through the string and checking if a substitution, an insertion, or a deletion needs to take place.
    sub = levenshtein(s1, string_length1 - 1, s2, string_length2 - 1);
    insert = levenshtein(s1, string_length1, s2, string_length2 - 1);
    del = levenshtein(s1, string_length1 - 1, s2, string_length2);

    // Check which method is superior
    if (sub > insert)
    {
        sub = insert;
    }
    if (sub > del)
    {
        sub = del;
    }

    // Return plus 1 to account for the last action performed
    return sub + 1;
}
